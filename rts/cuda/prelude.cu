
#include <hip/hip_runtime.h>
// start of prelude.cu

#define SCALAR_FUN_ATTR __device__ static inline
#define FUTHARK_FUN_ATTR __device__ static
#define FUTHARK_F64_ENABLED

typedef __int8_t int8_t;
typedef __int16_t int16_t;
typedef __int32_t int32_t;
typedef __int64_t int64_t;
typedef __uint8_t uint8_t;
typedef __uint16_t uint16_t;
typedef __uint32_t uint32_t;
typedef __uint64_t uint64_t;

#define __global
#define __local
#define __private
#define __constant
#define __write_only
#define __read_only

static inline __device__ int get_tblock_id(int d) {
  switch (d) {
  case 0: return blockIdx.x;
  case 1: return blockIdx.y;
  case 2: return blockIdx.z;
  default: return 0;
  }
}

static inline __device__ int get_num_tblocks(int d) {
  switch(d) {
  case 0: return gridDim.x;
  case 1: return gridDim.y;
  case 2: return gridDim.z;
  default: return 0;
  }
}

static inline __device__ int get_global_id(int d) {
  switch (d) {
    case 0: return threadIdx.x + blockIdx.x * blockDim.x;
    case 1: return threadIdx.y + blockIdx.y * blockDim.y;
    case 2: return threadIdx.z + blockIdx.z * blockDim.z;
    default: return 0;
  }
}

static inline __device__ int get_local_id(int d) {
  switch (d) {
    case 0: return threadIdx.x;
    case 1: return threadIdx.y;
    case 2: return threadIdx.z;
    default: return 0;
  }
}

static inline __device__ int get_local_size(int d) {
  switch (d) {
    case 0: return blockDim.x;
    case 1: return blockDim.y;
    case 2: return blockDim.z;
    default: return 0;
  }
}

static inline __device__ int get_global_size(int d) {
  switch (d) {
    case 0: return gridDim.x * blockDim.x;
    case 1: return gridDim.y * blockDim.y;
    case 2: return gridDim.z * blockDim.z;
    default: return 0;
  }
}


#define CLK_LOCAL_MEM_FENCE 1
#define CLK_GLOBAL_MEM_FENCE 2
static inline __device__ void barrier(int x) {
  __syncthreads();
}
static inline __device__ void mem_fence_local() {
  __threadfence_block();
}
static inline __device__ void mem_fence_global() {
  __threadfence();
}

static inline __device__ void barrier_local() {
  __syncthreads();
}

#ifndef NAN
#define NAN (0.0/0.0)
#endif
#ifndef INFINITY
#define INFINITY (1.0/0.0)
#endif
extern volatile __shared__ unsigned char shared_mem[];

#define SHARED_MEM_PARAM
#define FUTHARK_KERNEL extern "C" __global__ __launch_bounds__(MAX_THREADS_PER_BLOCK)
#define FUTHARK_KERNEL_SIZED(a,b,c) extern "C" __global__ __launch_bounds__(a*b*c)

// End of prelude.cu
