#include "hip/hip_runtime.h"
// Start of prelude.cu

#define FUTHARK_CUDA
#define FUTHARK_F64_ENABLED

typedef char int8_t;
typedef short int16_t;
typedef int int32_t;
typedef long long int64_t;
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;
typedef uint8_t uchar;
typedef uint16_t ushort;
typedef uint32_t uint;
typedef uint64_t ulong;
#define __kernel extern "C" __global__ __launch_bounds__(MAX_THREADS_PER_BLOCK)
#define __global
#define __local
#define __private
#define __constant
#define __write_only
#define __read_only

static inline int get_group_id(int d) {
  switch (d) {
  case 0: return blockIdx.x;
  case 1: return blockIdx.y;
  case 2: return blockIdx.z;
  default: return 0;
  }
}

static inline int get_num_groups(int d) {
  switch(d) {
  case 0: return gridDim.x;
  case 1: return gridDim.y;
  case 2: return gridDim.z;
  default: return 0;
  }
}

static inline int get_global_id(int d) {
  switch (d) {
    case 0: return threadIdx.x + blockIdx.x * blockDim.x;
    case 1: return threadIdx.y + blockIdx.y * blockDim.y;
    case 2: return threadIdx.z + blockIdx.z * blockDim.z;
    default: return 0;
  }
}

static inline int get_local_id(int d) {
  switch (d) {
    case 0: return threadIdx.x;
    case 1: return threadIdx.y;
    case 2: return threadIdx.z;
    default: return 0;
  }
}

static inline int get_local_size(int d) {
  switch (d) {
    case 0: return blockDim.x;
    case 1: return blockDim.y;
    case 2: return blockDim.z;
    default: return 0;
  }
}

static inline int get_global_size(int d) {
  switch (d) {
    case 0: return gridDim.x * blockDim.x;
    case 1: return gridDim.y * blockDim.y;
    case 2: return gridDim.z * blockDim.z;
    default: return 0;
  }
}


#define CLK_LOCAL_MEM_FENCE 1
#define CLK_GLOBAL_MEM_FENCE 2
static inline void barrier(int x) {
  __syncthreads();
}
static inline void mem_fence_local() {
  __threadfence_block();
}
static inline void mem_fence_global() {
  __threadfence();
}

static inline void barrier_local() {
  __syncthreads();
}

#define NAN (0.0/0.0)
#define INFINITY (1.0/0.0)
extern volatile __shared__ unsigned char local_mem[];

#define LOCAL_MEM_PARAM
#define REQD_GROUP_SIZE(a,b,c)

// End of prelude.cu
