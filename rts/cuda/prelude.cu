
#include <hip/hip_runtime.h>
// start of prelude.cu

#define SCALAR_FUN_ATTR __device__ static inline
#define FUTHARK_FUN_ATTR __device__ static
#define FUTHARK_F64_ENABLED

#if defined(__CUDACC_RTC__) || defined(__HIPCC_RTC__)
typedef char int8_t;
typedef short int16_t;
typedef int int32_t;
typedef long long int64_t;
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;
#else
// This is for the benefit of offline compilation with clang.
typedef signed char int8_t;
typedef short int16_t;
typedef int int32_t;
typedef long int64_t;
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long uint64_t;
#endif

#define __global
#define __local
#define __private
#define __constant
#define __write_only
#define __read_only

static inline __device__ int get_tblock_id(int d) {
  switch (d) {
  case 0: return blockIdx.x;
  case 1: return blockIdx.y;
  case 2: return blockIdx.z;
  default: return 0;
  }
}

static inline __device__ int get_num_tblocks(int d) {
  switch(d) {
  case 0: return gridDim.x;
  case 1: return gridDim.y;
  case 2: return gridDim.z;
  default: return 0;
  }
}

static inline __device__ int get_global_id(int d) {
  switch (d) {
    case 0: return threadIdx.x + blockIdx.x * blockDim.x;
    case 1: return threadIdx.y + blockIdx.y * blockDim.y;
    case 2: return threadIdx.z + blockIdx.z * blockDim.z;
    default: return 0;
  }
}

static inline __device__ int get_local_id(int d) {
  switch (d) {
    case 0: return threadIdx.x;
    case 1: return threadIdx.y;
    case 2: return threadIdx.z;
    default: return 0;
  }
}

static inline __device__ int get_local_size(int d) {
  switch (d) {
    case 0: return blockDim.x;
    case 1: return blockDim.y;
    case 2: return blockDim.z;
    default: return 0;
  }
}

static inline __device__ int get_global_size(int d) {
  switch (d) {
    case 0: return gridDim.x * blockDim.x;
    case 1: return gridDim.y * blockDim.y;
    case 2: return gridDim.z * blockDim.z;
    default: return 0;
  }
}


#define CLK_LOCAL_MEM_FENCE 1
#define CLK_GLOBAL_MEM_FENCE 2
static inline __device__ void barrier(int x) {
  __syncthreads();
}
static inline __device__ void mem_fence_local() {
  __threadfence_block();
}
static inline __device__ void mem_fence_global() {
  __threadfence();
}

static inline __device__ void barrier_local() {
  __syncthreads();
}

#if defined(__CUDACC_RTC__) || defined(__HIPCC_RTC__)
#define NAN (0.0/0.0)
#define INFINITY (1.0/0.0)
#endif

extern volatile __shared__ unsigned char shared_mem[];

#define SHARED_MEM_PARAM
#define FUTHARK_KERNEL extern "C" __global__ __launch_bounds__(MAX_THREADS_PER_BLOCK)
#define FUTHARK_KERNEL_SIZED(a,b,c) extern "C" __global__ __launch_bounds__(a*b*c)

// End of prelude.cu
