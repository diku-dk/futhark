
#include <hip/hip_runtime.h>
// Start of int_types.cu

#if defined(__CUDACC_RTC__) || defined(__HIPCC_RTC__)
typedef char int8_t;
typedef short int16_t;
typedef int int32_t;
typedef long long int64_t;
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;
#else
// This is for the benefit of offline compilation with clang.
typedef signed char int8_t;
typedef short int16_t;
typedef int int32_t;
typedef long int64_t;
typedef unsigned char uint8_t;
typedef unsigned short uint16_t;
typedef unsigned int uint32_t;
typedef unsigned long uint64_t;
#endif

// End of int_types.cu
